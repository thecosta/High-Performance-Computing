// Apply Sobel mask to grayed image. 
// by Bruno Costa Rendon


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
 
#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);


#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \


#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);

#define TILE_SIZE 3
#define CUDA_TIMING

unsigned char *input_gpu;
unsigned char *output_gpu;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        exit(-1);
    }
    #endif
    return result;
}

// GPU kernel and functions
__global__ void kernel(unsigned char *input,
                       unsigned char *output,
//                       unsigned char *mask_x,
//                       unsigned char *mask_y,
                       unsigned int height,
                       unsigned int width) {

    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;
    int index = x + y * width;
    
    int sobel_x[3][3] = {{1, 0, -1}, {2, 0, -2}, {1, 0, -1}};
    int sobel_y[3][3] = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};
    
    if (x < width-1 && y < height-1  && x > 0 && y > 0) {
        int sum_x = 0;
        int sum_y = 0;
        //printf("(1) sum_x = %d, sum_y = %d\n", sum_x, sum_y); 
//        printf("%d\n", mask_x[0]);
        for(int i = -1; i <= 1; i++) {
            for(int j = -1; j <=1; j++) {
                sum_x += input[(x+j) + (y+i)*width] * sobel_x[i+1][j+1];
                sum_y += input[(x+j) + (y+i)*width] * sobel_y[i+1][j+1];
            }
        }
//        printf("sum_x = %d, sum_y = %d\n", sum_x, sum_y);
        output[index] = sqrtf(sum_x*sum_x + sum_y*sum_y);
    //    printf("in loop");
    //printf("output[%d] = ", x*width+y);
    }
    __syncthreads();
}

void sobel_img(unsigned char *int_mat, 
	       unsigned char *out_mat, 
//               unsigned char *mask_x,
//               unsigned char *mask_y,
	       unsigned int height, 
	       unsigned int width) {
    printf("w = %d, h = %d\n", width, height);
    int gridXSize = 1 + (( width - 1) / TILE_SIZE);
    int gridYSize = 1 + ((height - 1) / TILE_SIZE);

    int XSize = gridXSize*TILE_SIZE;
    int YSize = gridYSize*TILE_SIZE;

    // Both are the same size (CPU/GPU).
    int size = XSize*YSize;

    // Allocate arrays in GPU memory
    checkCuda(hipMalloc((void**)&input_gpu, size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&output_gpu, size*sizeof(unsigned char)));

    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));

    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu,
                        int_mat,
                        height*width*sizeof(char),
                        hipMemcpyHostToDevice));
    
    // Wait for all threads to synchronize
    checkCuda(hipDeviceSynchronize());

     // Execute algorithm
    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

    #if defined(CUDA_TIMING)
        float Ktime;
        TIMER_CREATE(Ktime);
        TIMER_START(Ktime);
    #endif

    // Kernel Call
    kernel<<<dimGrid, dimBlock>>>(input_gpu, output_gpu, height, width);

    checkCuda(hipDeviceSynchronize());

    #if defined(CUDA_TIMING)
        TIMER_END(Ktime);
        printf("Kernel Execution Time: %f ms\n", Ktime);
    #endif

    // Retrieve results from the GPU
    checkCuda(hipMemcpy(out_mat,
                        output_gpu,
                        height*width*sizeof(unsigned char),
                        hipMemcpyDeviceToHost));

    // Free resources and end the program
    checkCuda(hipFree(output_gpu));
    checkCuda(hipFree(input_gpu));    
}
