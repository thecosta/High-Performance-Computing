// Find pixels within histogram range specified by user. 
// Add to gray color's count value atomically, and filter 
// out pixels not within histogram range. 
// by Bruno Costa Rendon

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
 
#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);


#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \


#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);

#define TILE_SIZE 3
#define CUDA_TIMING

unsigned char *input_gpu;
unsigned char *output_gpu;
int *out_histogram;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        exit(-1);
    }
    #endif
    return result;
}

// GPU kernel and functions
__global__ void kernel(unsigned char *input,
                       unsigned char *hist_image,
                       int* histogram,
                       int lower, int upper,
                       unsigned int height,
                       unsigned int width) {

    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;
    int index = x + y * width;

    if (x < width && y < height) {
        // If pixel is within histogram range. 
        if(input[index] >= lower && input[index] <= upper) {
            atomicAdd(&histogram[input[index]-lower], 1);
            hist_image[index] = input[index];
        }
        else 
            hist_image[index] = 0;
    }

    __syncthreads();
}

void classify(unsigned char *int_mat, 
	       unsigned char *hist_image, 
               int* histogram,
               int lower, int upper,
	       unsigned int height, 
	       unsigned int width) {
    int gridXSize = 1 + (( width - 1) / TILE_SIZE);
    int gridYSize = 1 + ((height - 1) / TILE_SIZE);

    int XSize = gridXSize*TILE_SIZE;
    int YSize = gridYSize*TILE_SIZE;

    // Both are the same size (CPU/GPU).
    int size = XSize*YSize;
    int hist_size = upper-lower+1;

    // Allocate arrays in GPU memory
    checkCuda(hipMalloc((void**)&input_gpu, size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&output_gpu, size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&out_histogram, hist_size*sizeof(int)));

    checkCuda(hipMemset(output_gpu, 0, size*sizeof(unsigned char)));
    checkCuda(hipMemset(out_histogram, 0, hist_size*sizeof(int)));

    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu,
                        int_mat,
                        height*width*sizeof(char),
                        hipMemcpyHostToDevice));
    
    // Wait for all threads to synchronize
    checkCuda(hipDeviceSynchronize());

     // Execute algorithm
    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

    printf("All memory allocated and set.\n");    

    #if defined(CUDA_TIMING)
        float Ktime;
        TIMER_CREATE(Ktime);
        TIMER_START(Ktime);
    #endif

    // Kernel Call
    kernel<<<dimGrid, dimBlock>>>(input_gpu, output_gpu, out_histogram, lower, upper, height, width);

    checkCuda(hipDeviceSynchronize());

    #if defined(CUDA_TIMING)
        TIMER_END(Ktime);
        printf("Kernel Execution Time: %f ms\n", Ktime);
    #endif

    // Retrieve results from the GPU
    checkCuda(hipMemcpy(hist_image,
                        output_gpu,
                        height*width*sizeof(unsigned char),
                        hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(histogram, 
                        out_histogram, 
                        hist_size*sizeof(int), 
                        hipMemcpyDeviceToHost));

    // Free resources and end the program
    checkCuda(hipFree(output_gpu));
    checkCuda(hipFree(input_gpu));
    checkCuda(hipFree(out_histogram));    
}
